#include "memory.cuh"

namespace gsg::cuda {

size_t move_to_device(const graph& input, int** device)
{
    size_t height = input.size;
    size_t width = input.size * sizeof(int);
    size_t pitch;

    HANDLE_ERROR(hipMallocPitch(device, &pitch, width, height));

    HANDLE_ERROR(hipMemcpy2D(*device, pitch, &input.matrix[0][0], width, width, height, hipMemcpyHostToDevice));

    return pitch;
}

void move_from_device(graph& input, int* device, size_t pitch)
{
    size_t height = input.size;
    size_t width = height * sizeof(int);

    HANDLE_ERROR(hipMemcpy2D(&input.matrix[0][0], width, device, pitch, width, height, hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(device));
}
}
