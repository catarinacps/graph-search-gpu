#include "hip/hip_runtime.h"
#include "device.cuh"

namespace gsg {

namespace cuda {

    static __global__ void fw_kernel(
        int const u,
        size_t pitch,
        int const n_vertex,
        int* const graph)
    {
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (y < n_vertex && x < n_vertex) {
            int indexYX = y * pitch + x;
            int indexUX = u * pitch + x;

            int new_path = graph[y * pitch + u] + graph[indexUX];
            int old_path = graph[indexYX];

            if (old_path > new_path)
                graph[indexYX] = new_path;
        }
    }

    static __global__ void bfs_kernel(
        node* Va, // nodes
        uint* Ea, // edges
        bool* Fa, // frontier
        bool* Xa, // visited
        int* Ca, // cost
        uint* num_nodes,
        bool* done)
    {
        int id = threadIdx.x + blockIdx.x * blockDim.x;

        if (id >= *num_nodes) {
            return;
        } else if (Fa[id] == true && Xa[id] == false) {
            Fa[id] = false;
            Xa[id] = true;
            __syncthreads();
            uint start = Va[id].first;
            uint end = start + Va[id].second;
            for (uint i = start; i < end; i++) {
                int nid = Ea[i];

                if (Xa[nid] == false) {
                    Ca[nid] = Ca[id] + 1;
                    Fa[nid] = true;
                    *done = false;
                }
            }
        }
    }

    bool bfs(const graph& input, uint searched_vertex, uint initial_vertex, uint block_size, bool verbose)
    {
        HANDLE_ERROR(hipSetDevice(0));

        uint num_blocks = (uint)ceil(input.size / (double)block_size);

        node* h_nodes = (node*)calloc(input.size, sizeof(node));
        uint* h_edges = (uint*)calloc(2 * input.num_edges, sizeof(uint));
        bool* h_frontier = (bool*)calloc(input.size, sizeof(bool));
        bool* h_visited = (bool*)calloc(input.size, sizeof(bool));
        int* h_cost = (int*)calloc(input.size, sizeof(int));

        h_frontier[initial_vertex] = true;

        uint edge_index = 0;
        for (uint i = 0; i < input.size; i++) {
            uint connected_edges = 0;
            for (uint j = 0; j < input.size; j++) {
                if (input.matrix[i][j] != 0) {
                    connected_edges++;
                    h_edges[edge_index++] = j;
                }
            }

            h_nodes[i].first = i;
            h_nodes[i].second = connected_edges;
            h_frontier[i] = false;
            h_visited[i] = false;
        }

        for (uint i = 0; i < input.size; i++)
            if (input.matrix[initial_vertex][i] != 0)
                h_frontier[i] = true;

        node* Va;
        HANDLE_ERROR(hipMalloc((void**)&Va, sizeof(node) * input.size));
        HANDLE_ERROR(hipMemcpy(Va, h_nodes, sizeof(node) * input.size, hipMemcpyHostToDevice));
        uint* Ea;
        HANDLE_ERROR(hipMalloc((void**)&Ea, sizeof(uint) * 2 * input.num_edges));
        HANDLE_ERROR(hipMemcpy(Ea, h_edges, sizeof(uint) * 2 * input.num_edges, hipMemcpyHostToDevice));
        bool* Fa;
        HANDLE_ERROR(hipMalloc((void**)&Fa, sizeof(bool) * input.size));
        HANDLE_ERROR(hipMemcpy(Fa, h_frontier, sizeof(bool) * input.size, hipMemcpyHostToDevice));
        bool* Xa;
        HANDLE_ERROR(hipMalloc((void**)&Xa, sizeof(bool) * input.size));
        HANDLE_ERROR(hipMemcpy(Xa, h_visited, sizeof(bool) * input.size, hipMemcpyHostToDevice));
        int* Ca;
        HANDLE_ERROR(hipMalloc((void**)&Ca, sizeof(int) * input.size));
        HANDLE_ERROR(hipMemcpy(Ca, h_cost, sizeof(int) * input.size, hipMemcpyHostToDevice));

        bool* d_over;
        HANDLE_ERROR(hipMalloc((void**)&d_over, sizeof(bool)));

        uint* d_num_nodes;
        HANDLE_ERROR(hipMalloc((void**)&d_num_nodes, sizeof(uint)));
        HANDLE_ERROR(hipMemcpy(d_num_nodes, &input.size, sizeof(uint), hipMemcpyHostToDevice));

        auto initial_time = get_time();

        int k = 0;
        bool stop;

        do {
            stop = true;

            HANDLE_ERROR(hipMemcpy(d_over, &stop, sizeof(bool), hipMemcpyHostToDevice));
            bfs_kernel<<<num_blocks, block_size>>>(Va, Ea, Fa, Xa, Ca, d_num_nodes, d_over);
            HANDLE_ERROR(hipMemcpy(&stop, d_over, sizeof(bool), hipMemcpyDeviceToHost));

            k++;
        } while (!stop);

        HANDLE_ERROR(hipGetLastError());
        HANDLE_ERROR(hipDeviceSynchronize());

        fmt::print("time: {}", get_time() - initial_time);

        HANDLE_ERROR(hipMemcpy(h_cost, Ca, sizeof(int) * input.size, hipMemcpyDeviceToHost));

        if (verbose) {
            fmt::print("\nnumber of times the kernel is called : {}\n", k);

            fmt::print("\ncost:\n");
            for (uint i = 0; i < input.size; i++)
                fmt::print("{} ", h_cost[i]);
        }

        free(h_nodes);
        free(h_edges);
        free(h_frontier);
        free(h_visited);
        free(h_cost);
        HANDLE_ERROR(hipFree(Va));
        HANDLE_ERROR(hipFree(Ea));
        HANDLE_ERROR(hipFree(Fa));
        HANDLE_ERROR(hipFree(Xa));
        HANDLE_ERROR(hipFree(Ca));
        HANDLE_ERROR(hipFree(d_over));

        return true;
    }

    bool floyd_warshall(const graph& input_host, uint block_size, bool verbose)
    {
        // int num_gpus;
        // HANDLE_ERROR(hipGetDeviceCount(&num_gpus));
        HANDLE_ERROR(hipSetDevice(0));

        auto n_vertex = input_host.size;

        dim3 dim_grid((n_vertex - 1) / block_size + 1, (n_vertex - 1) / block_size + 1, 1);
        dim3 dim_block(block_size, block_size, 1);

        int* d_matrix;
        auto pitch = move_to_device(input_host, &d_matrix);

        auto initial_time = get_time();

        hipFuncSetCacheConfig(reinterpret_cast<const void*>(fw_kernel), hipFuncCachePreferL1);

        for (int vertex = 0; vertex < n_vertex; ++vertex) {
            fw_kernel<<<dim_grid, dim_block>>>(vertex, pitch / sizeof(int), n_vertex, d_matrix);
        }

        HANDLE_ERROR(hipGetLastError());
        HANDLE_ERROR(hipDeviceSynchronize());

        fmt::print("time: {}", get_time() - initial_time);

        graph ret_graph(input_host.size);
        move_from_device(ret_graph, d_matrix, pitch);

        hipFree(d_matrix);

        return true;
    }
}

}
