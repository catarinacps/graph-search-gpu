#include "hip/hip_runtime.h"
#include "device.cuh"

namespace gsg {

namespace cuda {

    static __global__ void fw_kernel(
        int const u,
        size_t pitch,
        int const n_vertex,
        int* const graph)
    {
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (y < n_vertex && x < n_vertex) {
            int indexYX = y * pitch + x;
            int indexUX = u * pitch + x;

            int new_path = graph[y * pitch + u] + graph[indexUX];
            int old_path = graph[indexYX];

            if (old_path > new_path)
                graph[indexYX] = new_path;
        }
    }

    static __global__ void bfs_kernel(
        node* Va, // nodes
        int* Ea, // edges
        bool* Fa, // frontier
        bool* Xa, // visited
        int* Ca, // cost
        int num_nodes,
        bool* done)
    {
        int id = threadIdx.x + blockIdx.x * blockDim.x;

        if (id >= num_nodes) {
            *done = false;
        } else if (Fa[id] == true && Xa[id] == false) {
            printf("%d ", id); //This printf gives the order of vertices in BFS
            Fa[id] = false;
            Xa[id] = true;
            __syncthreads();
            int start = Va[id].first;
            int end = start + Va[id].second;
            for (int i = start; i < end; i++) {
                int nid = Ea[i];

                if (Xa[nid] == false) {
                    Ca[nid] = Ca[id] + 1;
                    Fa[nid] = true;
                    *done = false;
                }
            }
        }
    }

    bool bfs(const graph& input, uint searched_vertex, uint initial_vertex, uint block_size, bool verbose)
    {
        HANDLE_ERROR(hipSetDevice(0));

        int num_blocks = (int)ceil(input.size / (double)block_size);

        node* h_nodes = (node*)calloc(input.size, sizeof(node));
        int* h_edges = (int*)calloc(input.num_edges, sizeof(int));
        bool* h_frontier = (bool*)calloc(input.size, sizeof(bool));
        bool* h_visited = (bool*)calloc(input.size, sizeof(bool));
        int* h_cost = (int*)calloc(input.size, sizeof(int));

        h_frontier[initial_vertex] = true;

        node* Va;
        HANDLE_ERROR(hipMalloc((void**)&Va, sizeof(node) * input.size));
        HANDLE_ERROR(hipMemcpy(Va, h_nodes, sizeof(node) * input.size, hipMemcpyHostToDevice));
        int* Ea;
        HANDLE_ERROR(hipMalloc((void**)&Ea, sizeof(node) * input.size));
        HANDLE_ERROR(hipMemcpy(Ea, h_edges, sizeof(node) * input.size, hipMemcpyHostToDevice));
        bool* Fa;
        HANDLE_ERROR(hipMalloc((void**)&Fa, sizeof(bool) * input.size));
        HANDLE_ERROR(hipMemcpy(Fa, h_frontier, sizeof(bool) * input.size, hipMemcpyHostToDevice));
        bool* Xa;
        HANDLE_ERROR(hipMalloc((void**)&Xa, sizeof(bool) * input.size));
        HANDLE_ERROR(hipMemcpy(Xa, h_visited, sizeof(bool) * input.size, hipMemcpyHostToDevice));
        int* Ca;
        HANDLE_ERROR(hipMalloc((void**)&Ca, sizeof(int) * input.size));
        HANDLE_ERROR(hipMemcpy(Ca, h_cost, sizeof(int) * input.size, hipMemcpyHostToDevice));

        uint edge_index = 0;
        for (uint i = 0; i < input.size; i++) {
            uint connected_edges = 0;
            for (uint j = 0; j < input.size; j++) {
                if (input.matrix[i][j] != 0) {
                    connected_edges++;
                    h_edges[edge_index++] = j;
                }
            }

            h_nodes[i].first = i;
            h_nodes[i].second = connected_edges;
            h_frontier[i] = false;
            h_visited[i] = false;
        }

        bool* d_over;
        HANDLE_ERROR(hipMalloc((void**)&d_over, sizeof(bool)));

        dim3 grid(num_blocks, 1, 1);
        dim3 threads(block_size, 1, 1);

        auto initial_time = get_time();

        int k = 0;
        bool stop;

        do {
            stop = true;

            HANDLE_ERROR(hipMemcpy(d_over, &stop, sizeof(bool), hipMemcpyHostToDevice));
            bfs_kernel<<<grid, threads>>>(Va, Ea, Fa, Xa, Ca, input.size, d_over);
            HANDLE_ERROR(hipMemcpy(&stop, d_over, sizeof(bool), hipMemcpyDeviceToHost));

            k++;
        } while (!stop);

        HANDLE_ERROR(hipGetLastError());
        HANDLE_ERROR(hipDeviceSynchronize());

        fmt::print("time: {}", get_time() - initial_time);

        free(h_nodes);
        free(h_edges);
        free(h_frontier);
        free(h_visited);
        free(h_cost);
        HANDLE_ERROR(hipFree(Va));
        HANDLE_ERROR(hipFree(Ea));
        HANDLE_ERROR(hipFree(Fa));
        HANDLE_ERROR(hipFree(Xa));
        HANDLE_ERROR(hipFree(Ca));
        HANDLE_ERROR(hipFree(d_over));

        return true;
    }

    bool floyd_warshall(const graph& input_host, uint block_size, bool verbose)
    {
        // int num_gpus;
        // HANDLE_ERROR(hipGetDeviceCount(&num_gpus));
        HANDLE_ERROR(hipSetDevice(0));

        auto n_vertex = input_host.size;

        dim3 dim_grid((n_vertex - 1) / block_size + 1, (n_vertex - 1) / block_size + 1, 1);
        dim3 dim_block(block_size, block_size, 1);

        int* d_matrix;
        auto pitch = move_to_device(input_host, &d_matrix);

        auto initial_time = get_time();

        hipFuncSetCacheConfig(reinterpret_cast<const void*>(fw_kernel), hipFuncCachePreferL1);

        for (int vertex = 0; vertex < n_vertex; ++vertex) {
            fw_kernel<<<dim_grid, dim_block>>>(vertex, pitch / sizeof(int), n_vertex, d_matrix);
        }

        HANDLE_ERROR(hipGetLastError());
        HANDLE_ERROR(hipDeviceSynchronize());

        fmt::print("time: {}", get_time() - initial_time);

        graph ret_graph(input_host.size);
        move_from_device(ret_graph, d_matrix, pitch);

        hipFree(d_matrix);

        return true;
    }
}

}
